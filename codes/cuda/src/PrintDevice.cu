
#include <hip/hip_runtime.h>
#ifdef ENABLE_CUDA

#include <cuda_runtime.h>
#include <iostream>
using namespace std;

void printDeviceProp(const cudaDeviceProp& prop)
{
    cout << "Device Name : " << prop.name << "\n";
    cout << "totalGlobalMem : " << prop.totalGlobalMem << "\n";
    cout << "sharedMemPerBlock " << prop.sharedMemPerBlock << "\n";
    cout << "regsPerBlock : " << prop.regsPerBlock << "\n";
    cout << "warpSize :" << prop.warpSize << "\n";
    cout << "memPitch : " << prop.memPitch << "\n";
    cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << "\n";
    cout << "maxThreadsDim[0 - 2] : " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << "\n";
    cout << "maxGridSize[0 - 2] " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << "\n";
    cout << "totalConstMem : " << prop.totalConstMem << "\n";
    cout << "major.minor : " << prop.major << "." << prop.minor << "\n";
    cout << "clockRate : " << prop.clockRate << "\n";
    cout << "textureAlignment :" << prop.textureAlignment << "\n";
    cout << "deviceOverlap : " << prop.deviceOverlap << "\n";
    cout << "multiProcessorCount : " << prop.multiProcessorCount << "\n";
}

bool InitCUDA()
{
    //used to count the device numbers
    int count;

    cudaGetDeviceCount(&count);
    if (count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    // find the device >= 1.X
    int i;
    for (i = 0; i < count; ++i) {
        cudaDeviceProp prop;
        if (cudaGetDeviceProperties(&prop, i) == cudaSuccess) {
            if (prop.major >= 1) {
                printDeviceProp(prop);
                break;
            }
        }
    }

    // if can't find the device
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    // set cuda device
    cudaSetDevice(i);

    return true;
}

#endif // ENABLE_CUDA
